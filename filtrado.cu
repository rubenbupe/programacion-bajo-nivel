#include "hip/hip_runtime.h"
#include "filtrado.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <thread>
#include <vector>
#include <mutex>



__global__ void aplicaFiltroBilinearGPU(float* data, float* bilFilter, int w, int h, int numChannels, float* dataOut){
    int thId = blockIdx.x * blockDim.x + threadIdx.x;
    if(thId >= w * h)
        return;

    int i = thId / w;
    int j = thId % w;
    

    for(int z = 0; z < numChannels; z++){
        dataOut[((i * w + j) * numChannels) + z] = 0.0f;
        for(int yf = -1; yf < 2; yf++){
            for(int xf = -1; xf < 2; xf++){
                int y = (i + yf) % h;
                int x = (j + xf) % w;
                dataOut[((i * w + j) * numChannels )+ z] += data[((y * w + x) * numChannels )+ z] * bilFilter[(yf + 1) * 3 + (xf + 1)];
            }
        }
    }
}

void aplicaFiltroBilinear(float* data, int numChannels, int w, int h, float* &dataOut){
    int dataSize = h * w;
    int colorSize;
    float *d_data;
    float *d_bilFilter;
    float *d_dataOut;

    float bilFilter[9] = {  1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 
                            1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 
                            1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f  };

    if(numChannels == 3){
        colorSize = sizeof(frgbColor_t);
    }else if(numChannels == 4){
        colorSize = sizeof(frgbaColor_t);
    }

    float *dataOutTmp = new float[w * h * colorSize / sizeof(float)];

    hipMalloc(&d_data, dataSize * colorSize);
    hipMalloc(&d_dataOut, dataSize * colorSize);
    hipMalloc(&d_bilFilter, sizeof(float) * 9);
 
    hipMemcpy(d_data, data, dataSize * colorSize, hipMemcpyHostToDevice);
    // No lo hacemos en la GPU porque son solo 9 floats
    hipMemcpy(d_bilFilter, bilFilter, sizeof(float) * 9, hipMemcpyHostToDevice);


    int blockSize = 512;
	int numBlocks = dataSize/blockSize;
    numBlocks ++;

    printf("%d %d %d", dataSize, blockSize, numBlocks);

    aplicaFiltroBilinearGPU<<<numBlocks, blockSize >>>(d_data, d_bilFilter, w, h, numChannels, d_dataOut);
	hipMemcpy(dataOutTmp, d_dataOut,  dataSize * colorSize , hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

    dataOut = (float*) dataOutTmp;

}



void charToFloat(unsigned char* data, rgbaByteColor_t* palette, int w, int h, float* &dataOut){

    if(palette == nullptr){
        dataOut = new float[w*h*3];
        for(int i = 0; i < w*h*3; i++){
            dataOut[i] = ((float) data[i])/256.0f;
        }
    } else {
        dataOut = new float[w*h*4];
        for(int i = 0; i < w*h; i++){
            dataOut[i*4  ] = ((float) palette[data[i]].r)/256.0f;
            dataOut[i*4+1] = ((float) palette[data[i]].g)/256.0f;
            dataOut[i*4+2] = ((float) palette[data[i]].b)/256.0f;
            dataOut[i*4+3] = ((float) palette[data[i]].a)/256.0f;
        }
    }
}



void floatToChar(float* data, int w, int h, int numChannels, unsigned char* &dataOut){
    dataOut = new unsigned char[w*h*3];
    int dataOutCount = 0;
    for(int i = 0; i < w*h*numChannels; i+=numChannels){
        dataOut[dataOutCount++] = (unsigned char)(256.0f * data[i]);
        dataOut[dataOutCount++] = (unsigned char)(256.0f * data[i+1]);
        dataOut[dataOutCount++] = (unsigned char)(256.0f * data[i+2]);
    }
}