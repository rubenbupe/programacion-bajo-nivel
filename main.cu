#include "hip/hip_runtime.h"
// catalan++17
#include "bmp.h"
#include "filtrado.h"
#include <iostream>
#include <time.h>

int main(int argc, char **argv){
    hipSetDevice(0);
    fileBMP_t *bmp = loadBMP("./entrada/FLAG_B24.bmp");

    std::cout << "width: " << bmp->attributes.width << " height: " << bmp->attributes.height << " bpp: " << bmp->attributes.bpp << '\n';

    writeBMP(bmp, "./salida/salida.bmp");

    fileBMP_t* bmp2 = createBMP(bmp->attributes.width, bmp->attributes.height, bmp->attributes.bpp);
    bmp2->header = bmp->header;
    bmp2->attributes = bmp->attributes;

    float *data;
    float *dataOut;

    charToFloat(bmp->data, nullptr, bmp->attributes.width, bmp->attributes.height, data);

    clock_t begin = clock();
    aplicaFiltroBilinear(data, 3, bmp->attributes.width, bmp->attributes.height, dataOut);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    std::cout << "tiempo: " << time_spent << std::endl;

    floatToChar(dataOut, bmp2->attributes.width, bmp2->attributes.height, 3, bmp2->data);

    writeBMP(bmp2, "./salida/salida2.bmp");

    freeBMP(bmp);
    freeBMP(bmp2);
}